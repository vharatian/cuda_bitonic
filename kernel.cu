#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define ASC 1
#define DESC 2

#define DEBUGING 0
#define NOT_DEBUGING 1

#define MAX_BLOCK_DIM 65000

#define DEFAULT_SIZE 50000000

int execution_mod = NOT_DEBUGING;

int check_arguments(int argc, char** argv);
int fix_size(int size);
void bubble_sort(int* arr, int start, int end, int direction);
int generate_array(int** arr, int size);
void print_array(int* arr, int size);
void fill(int* arr, int start, int end, int value);
int checK_sort(int* arr, int size, int direction);
void print_error_message();

int up_devide(int devidend, int divisor);

void bitonic_sort(int* arr, int size, int direction);
__global__ void bitonic_device(int* arr, int size, int frame_size, int half_frame_size, int frame_assigned_count, int thread_assigned_size, int directione)
{
	if (frame_assigned_count > 1)
	{
		int block_start_element = (blockIdx.y * gridDim.x + blockIdx.x) * frame_size * frame_assigned_count;

		if (block_start_element < size)
		{
			if (block_start_element > size - (frame_size*frame_assigned_count))
				block_start_element = size - (frame_size*frame_assigned_count);

			int frame_number = threadIdx.x / half_frame_size;
			int first = block_start_element + frame_number * frame_size + (threadIdx.x % half_frame_size);
			int second = first + half_frame_size;

			if ((directione == ASC && arr[first] > arr[second]) || (directione == DESC && arr[first] < arr[second]))
			{
				int temp = arr[first];
				arr[first] = arr[second];
				arr[second] = temp;
			}
		}
	}
	else
	{
		int block_start_element = (blockIdx.y * gridDim.x + blockIdx.x) * frame_size;

		if (block_start_element < size)
		{
			if (block_start_element > size - frame_size)
				block_start_element = size - frame_size;

			int first = block_start_element + threadIdx.x * thread_assigned_size;
			int second = first + half_frame_size;

			if (threadIdx.x == blockDim.x - 1)
				thread_assigned_size = half_frame_size - (threadIdx.x  *  thread_assigned_size);

			int i;
			for (i = 0; i<thread_assigned_size; i++)
			{
				if ((directione == ASC && arr[first + i] > arr[second + i]) || (directione == DESC && arr[first + i] < arr[second + i]))
				{
					int temp = arr[first + i];
					arr[first + i] = arr[second + i];
					arr[second + i] = temp;
				}
			}
		}
	}
}

int main(int argc, char** argv)
{
	int size = check_arguments(argc, argv);
	if (size <= 0)
	{
		print_error_message();
		return 1;
	}


	int* arr;
	int fixed_size = generate_array(&arr, size);

	if (execution_mod == DEBUGING)
	{
		printf("=========== source array ============\n");
		print_array(arr, size);
		printf("\n");
	}

	printf("=========== execution started ============\n");
	time_t start_time = time(NULL);
	bitonic_sort(arr, fixed_size, ASC);
	time_t end_time = time(NULL);
	printf("=========== execution finished ============\n");

	if (execution_mod == DEBUGING)
	{
		printf("=========== result array ============\n\n");
		print_array(arr, size);
	}

	printf("\nexecution time : %d s\n\n", end_time - start_time);
	printf("result : %s\n", (checK_sort(arr, size, ASC))?"success":"failure");
	
    return 0;
}

int check_arguments(int argc, char** argv)
{
	if (argc == 1)
	{
		printf("default size selected : %d\n", DEFAULT_SIZE);
		return DEFAULT_SIZE;
	}
	else if (argc == 2)
	{
		return atoi(argv[1]);
	}
	else if (argc == 3)
	{
		if (argv[1][0] != '-' && argv[1][1] != 'd')
		{
			return -1;
		}
		else
			execution_mod = DEBUGING;
		
		return atoi(argv[2]);
	}
	else
	{
		return - 1;
	}
}

int generate_array(int** arr_ptr, int size)
{
	int fixed_size = fix_size(size);
	(*arr_ptr) = (int*)malloc(fixed_size*sizeof(int));

	int* arr = *arr_ptr;

	/*srand(time(NULL));
	int max = 3 * size;
	int i;
	for (i = 0; i < size; i++)
		arr[i] = rand() % max;

	fill(arr, size, fixed_size, INT_MAX);

	bubble_sort(arr, 0, size/2, DESC);
	bubble_sort(arr, size / 2, size, ASC);*/

	int i;
	int number = 0;
	for (i = 0; i < (size/2); i++)
	{
		arr[i] = number;
		number += 5;
	}

	for (; i < size; i++)
	{
		arr[i] = number;
		number -= 3;
	}

	return fixed_size;
}

void print_array(int* arr, int size)
{
	int i;

	printf("[");
	for (i = 0; i < size; i++)
	{
		printf(" %d", arr[i]);
	}

	printf(" ]\n");
}

void bubble_sort(int* arr, int start, int end, int direction)
{
	int i, j, temp;
	for (i = start; i < end; i++)
	{
		for (j = start; j < end - 1; j++)
		{
			if ((direction == ASC && arr[j] > arr[j+1]) || (direction == DESC &&  arr[j] < arr[j+1]))
			{
				temp = arr[j];
				arr[j] = arr[j+1];
				arr[j+1] = temp;
			}
		}
	}
}

void bitonic_sort(int* arr, int size, int direction)
{
	int* device_arr;
	hipError_t cuda_status;

	int selected_device = 0;
	struct hipDeviceProp_t properties;
	cuda_status = hipGetDeviceProperties(&properties, selected_device);
	if (cuda_status == hipSuccess)
	{
		cuda_status = hipSetDevice(selected_device);
		if (cuda_status == hipSuccess)
		{
			cuda_status = hipMalloc((void**)&device_arr, size * sizeof(int));
			if (cuda_status == hipSuccess)
			{
				cuda_status = hipMemcpy(device_arr, arr, size * sizeof(int), hipMemcpyHostToDevice);
				if (cuda_status == hipSuccess)
				{
					dim3 block_dim;
					dim3 thread_dim;

					int max_thread = properties.maxThreadsPerMultiProcessor / 6;
					int frame_size;
					int frame_assigned_count = 1;
					int thread_assigned_size;
					int block_x;

					for (frame_size = size; frame_size > 1; frame_size /= 2)
					{
						

						if (frame_size / 2 > max_thread)
						{
							thread_dim = max_thread;
							block_x = sqrt(1.0*(size / frame_size)) + 1;
							frame_assigned_count = 1;
							thread_assigned_size = (frame_size / (2 * thread_dim.x));
						}
						else
						{
							for (frame_assigned_count = 1; (frame_size / 2) * frame_assigned_count < max_thread && size / (frame_assigned_count * frame_size) > 200; frame_assigned_count++);
							//frame_assigned_count = 1;
							thread_dim = frame_assigned_count * (frame_size / 2);
							thread_assigned_size = 1;

							block_x = sqrt(1.0*(size / (frame_assigned_count * frame_size))) + 1;
						}
						block_dim = dim3(block_x, block_x);

						bitonic_device << <block_dim, thread_dim >> >(device_arr, size, frame_size, frame_size / 2, frame_assigned_count, thread_assigned_size, direction);

						if (execution_mod == DEBUGING)
						{
							printf("block_count : %d, frame_size : %d, farme_assinge_count : %d\n", block_dim.x*block_dim.y, frame_size, frame_assigned_count);
							hipMemcpy(arr, device_arr, size*sizeof(int), hipMemcpyDeviceToHost);
							print_array(arr, size);
						}

						cuda_status = hipGetLastError();
						if (cuda_status != hipSuccess)
						{
							fprintf(stderr, "faild to run kernel function, block count : %d, thread count : %d\n", block_dim.x, thread_dim.x);
							break;
						}
					}

					cuda_status = hipMemcpy(arr, device_arr, size*sizeof(int), hipMemcpyDeviceToHost);
					if (cuda_status != hipSuccess)
						fprintf(stderr, "faild to copy arrry from device to host, array size : %d\n" + size * sizeof(int));
				}
				else
					fprintf(stderr, "faild to copy array from host to device, array size : %d\n", size * sizeof(int));
			}
			else
				fprintf(stderr, "failed to allocate memory on device, requested memoy : %d\n", size * sizeof(int));
		}
		else
			fprintf(stderr, "failed to select cuda capable device, device num : %d\n", selected_device);
	}
	else
		fprintf(stderr, "failed to read device properies, device num : %d\n", selected_device);
	
	

	hipFree(device_arr);
}

int up_devide(int devidend, int divisor)
{
	int result = devidend / divisor;
	/*if (devidend % divisor != 0)
		result++;*/

	return result;
}

int checK_sort(int* arr, int size, int direction)
{
	int i;
	for (i = 0; i < size - 1; i++)
		if ((direction == ASC && arr[i] > arr[i + 1]) || (direction == DESC && arr[i] < arr[i + 1]))
			return false;

	return true;
}


int fix_size(int size)
{
	int result;
	for (result = 2; result < size; result *= 2);

	return result;
}

void fill(int* arr, int start, int end, int value)
{
	int i;
	for (i = start; i < end; i++)
		arr[i] = value;
}

void print_error_message()
{
	printf("wrong command format.\n betonic [option] [size] \n option : \n \t -d debug mod\n");
}
